
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>


typedef struct {
  int nfft;
  hipStream_t stream;
  hipfftHandle plan;
  float2 *d_in;
  float2 *d_out;
  float2 *h_in;
  float2 *h_out;
} fft_stream_struct;

void fft_stream_setup(fft_stream_struct *fss, int nfft) {
  (*fss).nfft = nfft;
  hipStreamCreate(&(*fss).stream);
  hipfftPlan1d(&(*fss).plan, nfft, HIPFFT_C2C, 1);
  hipfftSetStream((*fss).plan, (*fss).stream);
  hipMalloc((void**)&(*fss).d_in, nfft * sizeof(float2));
  hipMalloc((void**)&(*fss).d_out, nfft * sizeof(float2));
  hipHostAlloc((void**)&(*fss).h_in, sizeof(float2) * nfft, hipHostMallocPortable);
  hipHostAlloc((void**)&(*fss).h_out, sizeof(float2) * nfft, hipHostMallocPortable);
}

void fft_stream_destroy(fft_stream_struct *fss) {
  hipStreamDestroy((*fss).stream);
  hipfftDestroy((*fss).plan);
  hipFree((*fss).d_in);
  hipFree((*fss).d_out);
  hipHostUnregister((*fss).h_in);
  hipHostUnregister((*fss).h_out);
  hipHostFree((*fss).h_in);
  hipHostFree((*fss).h_out);
}

void fft_stream_push(fft_stream_struct *fss) {
  int nfft = (*fss).nfft;
  hipMemcpyAsync((*fss).d_in, (*fss).h_in, nfft * sizeof(float2), hipMemcpyHostToDevice, (*fss).stream);
  hipfftExecC2C((*fss).plan, (hipfftComplex*)(*fss).d_in, (hipfftComplex*)(*fss).d_out, HIPFFT_FORWARD);
  hipMemcpyAsync((*fss).h_out, (*fss).d_in, nfft * sizeof(float2), hipMemcpyDeviceToHost, (*fss).stream);
}

int main() {
    const int NUM_STREAMS = 3;
    const int NFFT = 1 << 25;

    // Creates fft streams
    fft_stream_struct streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
      fft_stream_setup(&streams[i], NFFT);
    }
    printf("Finished FFT stream setup\n");
    fflush(stdout);

    // Host input data initialization
    for (int j = 0; j < NUM_STREAMS; j++) {
      for (int i = 0; i < NFFT; i++) {
          streams[j].h_in[i].x = 1.f;
          streams[j].h_in[i].y = 0.f;
      }
    }
    printf("Finished populating input data\n");
    fflush(stdout);

    // Async memcopies and computations
    for (int i = 0; i < 3 * NUM_STREAMS; i++) {
      fft_stream_push(&streams[i % NUM_STREAMS]);
    }
    printf("Finished FFTs\n");
    fflush(stdout);

    for (int i = 0; i < NUM_STREAMS; i++) {
      hipStreamSynchronize(streams[i].stream);
    }
    printf("Finished synchronizing\n");
    fflush(stdout);

    hipDeviceSynchronize();

    for (int i = 0; i < NUM_STREAMS; i++) {
      fft_stream_destroy(&streams[i]);
    }
    printf("Finished cleaning up\n");
    fflush(stdout);

    hipDeviceReset();

    return 0;
}
