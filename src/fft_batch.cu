#define NX 1048576
#define BATCH 10
#define RANK 1


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>


int main() {

  hipfftHandle plan;
  hipfftComplex *data;
  hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return -1;
  }

  int dims[] = { NX };
  if (hipfftPlanMany(&plan, RANK, dims, NULL, 0, 0,
              NULL, 0, 0, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return -1;
  }

  /* Note:
   *  Identical pointers to input and output arrays implies in-place transformation
   */

  if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return -1;
  }

  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
    return -1;
  }

  /*
   *  Results may not be immediately available so block device until all
   *  tasks have completed
   */

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return -1;
  }

  /*
   *  Divide by number of elements in data set to get back original data
   */

  hipfftDestroy(plan);
  hipFree(data);

  return 0;
}
